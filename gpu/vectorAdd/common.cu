#include "common.h"

void printDeviceProperties()
{
	struct hipDeviceProp_t deviceProp;
	int ret = hipGetDeviceProperties(&deviceProp, 0);
	CPE(ret != hipSuccess, "Get Device Properties failed\n", -1);

	printf("Device name: %s\n", deviceProp.name);
	printf("Total global memory: %lu bytes\n", deviceProp.totalGlobalMem);
	printf("Warp size: %d\n", deviceProp.warpSize);
	printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);

	printf("Multi-processor count: %d\n", deviceProp.multiProcessorCount);
	printf("Threads per multi-processor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
}
