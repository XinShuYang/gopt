#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <unistd.h>

/**< NVCC assumes that all header files are C++ files. Tell it that these are
  *  C header files. */
extern "C" {
#include "ipv4.h"
#include "worker-master.h"
#include "util.h"
}

__global__ void
ipv4Gpu(uint32_t *req, uint16_t *resp, 
	uint16_t *tbl24, uint16_t *tbl8,
	int num_reqs)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < num_reqs) {
		uint32_t ip = req[i];
		uint32_t tbl24_index = (ip >> 8);
		uint16_t tbl_entry;

		/* Copy tbl24 entry */
		tbl_entry = tbl24[tbl24_index];

		/* Copy tbl8 entry (only if needed) */
		if((tbl_entry & RTE_LPM_VALID_EXT_ENTRY_BITMASK) ==
				RTE_LPM_VALID_EXT_ENTRY_BITMASK) {

			unsigned tbl8_index = (uint8_t) ip +
					((uint8_t) tbl_entry * RTE_LPM_TBL8_GROUP_NUM_ENTRIES);

			tbl_entry = tbl8[tbl8_index];
		}

		resp[i] = tbl_entry;
	}
}

/**< wmq: the worker/master queue for all lcores. Non-NULL iff the lcore is an
  *  active worker. */
void master_gpu(volatile struct wm_queue *wmq, hipStream_t my_stream,
	uint32_t *h_reqs, uint32_t *d_reqs,	/**< Kernel inputs */
	uint16_t *h_resps, uint16_t *d_resps,	/**< Kernel outputs */
	uint16_t *d_tbl24, uint16_t *d_tbl8,	/**< IPv4 lookup tables */
	int num_workers, int *worker_lcores)
{
	assert(num_workers != 0);
	assert(worker_lcores != NULL);
	
	int i, err;

	/**< Variables for batch-size and latency averaging measurements */
	int msr_iter = 0;			/**< Number of kernel launches */
	long long msr_tot_req = 0;	/**< Total packet serviced by the master */
	struct timespec msr_start, msr_end;
	double msr_tot_us = 0;		/**< Total microseconds over all iterations */

	/**< The GPU-buffer (h_reqs) start index for a worker's packets during a
	  *  kernel launch. */
	int req_lo[WM_MAX_LCORE] = {0};

	/**< Number of requests that we'll send to the GPU = nb_req. We don't need
	  *  to worry about nb_req overflowing the capacity of h_reqs because it
	  *  fits all WM_MAX_LCORE. */
	int nb_req = 0;

	/**<  Value of the queue-head from an lcore during the last iteration*/
	long long prev_head[WM_MAX_LCORE] = {0}, new_head[WM_MAX_LCORE] = {0};
	
	int w_i, w_lid;		/**< A worker-iterator and the worker's lcore-id */
	volatile struct wm_queue *lc_wmq;	/**< Work queue of one worker */

	clock_gettime(CLOCK_REALTIME, &msr_start);

	while(1) {

		/**< Copy all the requests supplied by workers into the contiguous 
		  *  h_reqs buffer. */
		for(w_i = 0; w_i < num_workers; w_i ++) {
			w_lid = worker_lcores[w_i];		/**< Don't use w_i after this */
			lc_wmq = &wmq[w_lid];
			
			/**< Snapshot this worker queue's head. The entries in the queue up
			  *  to index (lc_wmq->head - 1) are definitely valid. The entry at
			  *  index lc_wmq->head also might be valid in some cases - we will
			  *  process it in the next iteration */
			new_head[w_lid] = lc_wmq->head;

			/**< Record the beginning of the GPU req. buffer for this lcore */
			req_lo[w_lid] = nb_req;

			/**< Add the new packets from this lcore to the request buffer */
			for(i = prev_head[w_lid]; i < new_head[w_lid]; i ++) {
				int q_i = i & WM_QUEUE_CAP_;	/**< Queues are circular */
				uint32_t req = lc_wmq->reqs[q_i];

				h_reqs[nb_req] = req;
				nb_req ++;
			}
		}

		if(nb_req == 0) {	/**< No new packets from any worker? */
			continue;
		}

		/**< Copy requests to device */
		err = hipMemcpyAsync(d_reqs, h_reqs, nb_req * sizeof(uint32_t), 
			hipMemcpyHostToDevice, my_stream);
		CPE(err != hipSuccess, "Failed to copy requests h2d\n");

		/**< Kernel launch */
		int threadsPerBlock = 256;
		int blocksPerGrid = (nb_req + threadsPerBlock - 1) / threadsPerBlock;
	
		ipv4Gpu<<<blocksPerGrid, threadsPerBlock, 0, my_stream>>>(d_reqs, 
			d_resps, d_tbl24, d_tbl8, nb_req);
		err = hipGetLastError();
		CPE(err != hipSuccess, "Failed to launch ipv4Gpu kernel\n");

		/**< Copy responses from device */
		err = hipMemcpyAsync(h_resps, d_resps, nb_req * sizeof(uint16_t),
			hipMemcpyDeviceToHost, my_stream);
		CPE(err != hipSuccess, "Failed to copy responses d2h\n");

		/**< Synchronize all CUDA operations */
		hipStreamSynchronize(my_stream);
		
		/**< Copy the responses back to worker queues */
		for(w_i = 0; w_i < num_workers; w_i ++) {
			w_lid = worker_lcores[w_i];		/**< Don't use w_i after this */
			lc_wmq = &wmq[w_lid];

			for(i = prev_head[w_lid]; i < new_head[w_lid]; i ++) {
				/**< Offset in this workers' queue and the GPU req. buffer */
				int q_i = i & WM_QUEUE_CAP_;				
				int req_i = req_lo[w_lid] + (i - prev_head[w_lid]);
				lc_wmq->resps[q_i] = h_resps[req_i];
			}

			prev_head[w_lid] = new_head[w_lid];
		
			/**< Update tail for this worker */
			lc_wmq->tail = new_head[w_lid];
		}

		/**< Do some GPU-specific measurements */
		msr_iter ++;
		msr_tot_req += nb_req;

		if(msr_iter == 100000) {
			clock_gettime(CLOCK_REALTIME, &msr_end);
			msr_tot_us = (msr_end.tv_sec - msr_start.tv_sec) * 1000000 +
				(msr_end.tv_nsec - msr_start.tv_nsec) / 1000;

			blue_printf("\tGPU master: average batch size = %lld\n"
				"\t\tAverage time for GPU communication = %f us\n",
				msr_tot_req / msr_iter, msr_tot_us / msr_iter);

			msr_iter = 0;
			msr_tot_req = 0;

			/**< Start the next measurement */
			clock_gettime(CLOCK_REALTIME, &msr_start);
		}

		nb_req = 0;
	}
}

int main(int argc, char **argv)
{
	int c, i, err = hipSuccess;
	int lcore_mask = -1;
	hipStream_t my_stream;
	volatile struct wm_queue *wmq;

	/**< CUDA buffers */
	uint32_t *h_reqs, *d_reqs;
	uint16_t *h_resps, *d_resps;	
	uint16_t *d_tbl24, *d_tbl8;	/**< No need for host pinned memory */

	struct rte_lpm *lpm;

	/**< Get the worker lcore mask */
	while ((c = getopt (argc, argv, "c:")) != -1) {
		switch(c) {
			case 'c':
				/**< atoi() doesn't work for hex representation */
				lcore_mask = strtol(optarg, NULL, 16);
				break;
			default:
				blue_printf("\tGPU master: I need coremask. Exiting!\n");
				exit(-1);
		}
	}

	assert(lcore_mask != -1);
	blue_printf("\tGPU master: got lcore_mask: %x\n", lcore_mask);

	/**< Create a CUDA stream */
	err = hipStreamCreate(&my_stream);
	CPE(err != hipSuccess, "Failed to create cudaStream\n");

	/**< Allocate hugepages for the shared queues */
	blue_printf("\tGPU master: creating worker-master shm queues\n");
	int wm_queue_bytes = M_2;
	while(wm_queue_bytes < WM_MAX_LCORE * sizeof(struct wm_queue)) {
		wm_queue_bytes += M_2;
	}
	printf("\t\tTotal size of wm_queues = %d hugepages\n", 
		wm_queue_bytes / M_2);
	wmq = (volatile struct wm_queue *) hrd_malloc_socket(WM_QUEUE_KEY,
		wm_queue_bytes, 0);

	/**< Ensure that queue counters are in separate cachelines */
	for(i = 0; i < WM_MAX_LCORE; i ++) {
		uint64_t c1 = (uint64_t) (uintptr_t) &wmq[i].head;
		uint64_t c2 = (uint64_t) (uintptr_t) &wmq[i].tail;
		uint64_t c3 = (uint64_t) (uintptr_t) &wmq[i].sent;

		assert((c1 % 64 == 0) && (c2 % 64 == 0) && (c3 % 64 == 0));
	}

	blue_printf("\tGPU master: creating worker-master shm queues done\n");

	/**< Allocate buffers for requests from all workers*/
	blue_printf("\tGPU master: creating buffers for requests\n");
	int reqs_buf_size = WM_QUEUE_CAP * WM_MAX_LCORE * sizeof(uint32_t);
	err = hipHostMalloc((void **) &h_reqs, reqs_buf_size);
	CPE(err != hipSuccess, "Failed to hipHostMalloc req buffer\n");
	err = hipMalloc((void **) &d_reqs, reqs_buf_size);
	CPE(err != hipSuccess, "Failed to hipMalloc req buffer\n");

	/**< Allocate buffers for responses for all workers */
	blue_printf("\tGPU master: creating buffers for responses\n");
	int resps_buf_size = WM_QUEUE_CAP * WM_MAX_LCORE * sizeof(uint16_t);
	err = hipHostMalloc((void **) &h_resps, resps_buf_size);
	CPE(err != hipSuccess, "Failed to hipHostMalloc resp buffers\n");
	err = hipMalloc((void **) &d_resps, resps_buf_size);
	CPE(err != hipSuccess, "Failed to hipMalloc resp buffers\n");

	/**< Create the IPv4 cache and copy it over */
	blue_printf("\tGPU master: creating rte_lpm lookup table\n");
	lpm = ipv4_init(IPv4_PORT_MASK);

	/**< rte_lpm_tbl24_entry ~ rte_lpm_tbl8_entry ~ uint16_t */
	int entry_sz = sizeof(struct rte_lpm_tbl24_entry);
	int tbl24_bytes = RTE_LPM_TBL24_NUM_ENTRIES * entry_sz;
	int tbl8_bytes = RTE_LPM_TBL8_NUM_ENTRIES * entry_sz;
	
	/**< Alloc and copy tbl24 and tbl8 arrays to GPU memory */
	blue_printf("\tGPU master: alloc tbl24 (size = %d bytes) on device\n",
		tbl24_bytes);
	err = hipMalloc((void **) &d_tbl24, tbl24_bytes);
	CPE(err != hipSuccess, "Failed to hipMalloc tbl24\n");
	hipMemcpy(d_tbl24, lpm->tbl24, tbl24_bytes, hipMemcpyHostToDevice);

	blue_printf("\tGPU master: alloc tbl8 (size = %d bytes) on device\n",
		tbl8_bytes);
	err = hipMalloc((void **) &d_tbl8, tbl8_bytes);
	CPE(err != hipSuccess, "Failed to hipMalloc tbl8\n");
	hipMemcpy(d_tbl8, lpm->tbl8, tbl8_bytes, hipMemcpyHostToDevice);

	int num_workers = bitcount(lcore_mask);
	int *worker_lcores = get_active_bits(lcore_mask);
	
	/**< Launch the GPU master */
	blue_printf("\tGPU master: launching GPU code\n");
	master_gpu(wmq, my_stream,
		h_reqs, d_reqs, 
		h_resps, d_resps, 
		d_tbl24, d_tbl8,
		num_workers, worker_lcores);
	
}
