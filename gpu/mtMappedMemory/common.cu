#include "common.h"

void printDeviceProperties()
{
	struct hipDeviceProp_t deviceProp;
	int ret = hipGetDeviceProperties(&deviceProp, 0);
	CPE(ret != hipSuccess, "Get Device Properties failed\n", -1);

	printf("\n=================DEVICE PROPERTIES=================\n");
	printf("\tDevice name: %s\n", deviceProp.name);
	printf("\tTotal global memory: %lu bytes\n", deviceProp.totalGlobalMem);
	printf("\tWarp size: %d\n", deviceProp.warpSize);
	printf("\tCompute capability: %d.%d\n", deviceProp.major, deviceProp.minor);

	printf("\tMulti-processor count: %d\n", deviceProp.multiProcessorCount);
	printf("\tThreads per multi-processor: %d\n", deviceProp.maxThreadsPerMultiProcessor);

	printf("\n");
}

long long get_cycles()
{
	unsigned low, high;
	unsigned long long val;
	asm volatile ("rdtsc" : "=a" (low), "=d" (high));
	val = high;
	val = (val << 32) | low;
	return val;
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call.
hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %sn", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

// Returns when all N elements in A are non-zero
void waitForNonZero(volatile int *A, int N, int tid)
{
	int i, turns = 0;
	while(1) {
		int allNonZero = 1;
		int zeroAt = -1;
		for(i = 0; i < N; i ++) {
			if(A[i] == 0) {
				allNonZero = 0;
				zeroAt = i;
				break;
			}
		}

		if(allNonZero) {
			return;
		}

		turns ++;
		if(turns > 1000000000) {
			printf("Thread %d: Waiting for non-zero at index %d\n", tid, zeroAt);
			turns = 0;
		}
	}
}
