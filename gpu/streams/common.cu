#include "common.h"

void printDeviceProperties()
{
	struct hipDeviceProp_t deviceProp;
	int ret = hipGetDeviceProperties(&deviceProp, 0);
	CPE(ret != hipSuccess, "Get Device Properties failed\n", -1);

	printf("\n=================DEVICE PROPERTIES=================\n");
	printf("\tDevice name: %s\n", deviceProp.name);
	printf("\tTotal global memory: %lu bytes\n", deviceProp.totalGlobalMem);
	printf("\tWarp size: %d\n", deviceProp.warpSize);
	printf("\tCompute capability: %d.%d\n", deviceProp.major, deviceProp.minor);

	printf("\tMulti-processor count: %d\n", deviceProp.multiProcessorCount);
	printf("\tThreads per multi-processor: %d\n", deviceProp.maxThreadsPerMultiProcessor);

	printf("\n");
}

long long get_cycles()
{
	unsigned low, high;
	unsigned long long val;
	asm volatile ("rdtsc" : "=a" (low), "=d" (high));
	val = high;
	val = (val << 32) | low;
	return val;
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call.
hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %sn", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}
