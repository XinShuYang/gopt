#include "hip/hip_runtime.h"
#include "common.h"
#include <assert.h>
#include <sys/ipc.h>
#include <sys/shm.h>

hipStream_t myStream;

__global__ void
seqMem(const long long *log, long long *sum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j, num_iters = LOG_CAP / CUDA_THREADS;
	int iter_base;

	sum[i] = 0;

	for(j = 0; j < num_iters; j ++) {
		iter_base = j * CUDA_THREADS;	
		sum[i] += log[iter_base + i];
	}
}

double cpu_run(long long *log)
{
	int i;
	long long sum = 0;
	
	struct timespec start, end;
	clock_gettime(CLOCK_REALTIME, &start);

	for(i = 0; i < LOG_CAP; i ++) {
		sum += log[i];
	}

	clock_gettime(CLOCK_REALTIME, &end);

	printf("cpu_run: sum = %lld\n", sum);

	double time = (double) (end.tv_nsec - start.tv_nsec) / 1000000000 + 
		(end.tv_sec - start.tv_sec);
	return time;
}

double gpu_run(long long *d_log)
{
	struct timespec start, end;
	int err = hipSuccess;
	long long *h_sum, *d_sum, tot_sum = 0;

	int i;

	assert(LOG_CAP % CUDA_THREADS == 0);

	/**< Allocate a sum-buffer on the GPU */
	h_sum = (long long *) malloc(CUDA_THREADS * sizeof(long long));
	err = hipMalloc((void **) &d_sum, CUDA_THREADS * sizeof(long long));
	CPE(err != hipSuccess, "Failed to allocate sum buffer on GPU\n", -1);

	/**< Kernel launch */
	int threadsPerBlock = 256;
	int blocksPerGrid = CUDA_THREADS / threadsPerBlock;

	clock_gettime(CLOCK_REALTIME, &start);

	seqMem<<<blocksPerGrid, threadsPerBlock, 0, myStream>>>(d_log, d_sum);
	err = hipGetLastError();
	CPE(err != hipSuccess, "Failed to launch seqMem kernel\n", -1);

	/**< Wait for the kernel to complete */
	hipStreamSynchronize(myStream);

	clock_gettime(CLOCK_REALTIME, &end);

	/**< Copy back the sum buffer */
	err = hipMemcpyAsync(h_sum, d_sum, CUDA_THREADS * sizeof(long long),
		hipMemcpyDeviceToHost, myStream);
	CPE(err != hipSuccess, "Failed to copy C from device to host\n", -1);

	for(i = 0; i < CUDA_THREADS; i ++) {
		tot_sum += h_sum[i];
	}

	printf("gpu_run: sum = %lld\n", tot_sum);

	double time = (double) (end.tv_nsec - start.tv_nsec) / 1000000000 + 
		(end.tv_sec - start.tv_sec);
	
	return time;
}

int main(int argc, char *argv[])
{
	int err = hipSuccess;
	int i;
	long long *h_log, *d_log;

	srand(time(NULL));

	printDeviceProperties();

	/** <Initialize a cudaStream for async calls */
	err = hipStreamCreate(&myStream);
	CPE(err != hipSuccess, "Failed to create cudaStream\n", -1);

	printf("Creating log of size %lu bytes\n", LOG_CAP * sizeof(long long));
	/** <Initialize hugepage log and copy it to the device: do it once */
#if USE_HUGEPAGE == 1
	int sid = shmget(1, LOG_CAP * sizeof(long long), SHM_HUGETLB | 0666 | IPC_CREAT);
	assert(sid >= 0);
	h_log = (long long *) shmat(sid, 0, 0);
#else
	h_log = (long long *) malloc(LOG_CAP * sizeof(long long));
#endif
	assert(h_log != NULL);

	for(i = 0; i < LOG_CAP; i ++) {
		h_log[i] = i;
	}
	err = hipMalloc((void **) &d_log, LOG_CAP * sizeof(long long));
	CPE(err != hipSuccess, "Failed to allocate log on device\n", -1);

	printf("Copying log to device\n");
	err = hipMemcpy(d_log, h_log, LOG_CAP * sizeof(long long), hipMemcpyHostToDevice);
	CPE(err != hipSuccess, "Failed to copy to device memory\n", -1);

	double cpu_time, gpu_time;

	cpu_time = cpu_run(h_log);
	gpu_time = gpu_run(d_log);
	gpu_time = gpu_run(d_log) + gpu_time / 10000000000;
	
	printf("CPU: time = %f, %d GB/s\n", cpu_time,
		(int) ((LOG_CAP * sizeof(long long)) / (cpu_time * 1000000000)));
	printf("GPU: time = %f, %d GB/s\n", gpu_time,
		(int) ((LOG_CAP * sizeof(long long)) / (gpu_time * 1000000000)));
	printf("\n");
	
	// Free device memory
	hipFree(d_log);

#if USE_HUGEPAGE == 0
	free(h_log);
#endif

	// Reset the device and exit
	err = hipDeviceReset();
	CPE(err != hipSuccess, "Failed to de-initialize the device\n", -1);

	printf("Done\n");
	return 0;
}

