#include "hip/hip_runtime.h"
#include "common.h"

#define G_1 1000000000
__global__ void
vectorAdd(int *A, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
		A[i] *= A[i];
	}
}

int cmpfunc (const void *a, const void *b)
{
	double a_d = *(double *) a;
	double b_d = *(double *) b;

	if(a_d > b_d) {
		return 1;
	} else if(a_d < b_d) {
		return -1;
	} else {
		return 0;
	}
}

void dummy_run(int *h_A, int *d_A, int num_pkts, hipStream_t my_stream)
{
	int err = hipSuccess;
	int threadsPerBlock = 256;
	int blocksPerGrid = (num_pkts + threadsPerBlock - 1) / threadsPerBlock;

	err = hipMemcpyAsync(d_A, h_A, num_pkts * sizeof(int), 
		hipMemcpyHostToDevice, my_stream);
	CPE(err != hipSuccess, "H2D memcpy failed\n");
	
	vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, my_stream>>>(d_A, num_pkts);
	err = hipGetLastError();
	CPE(err != hipSuccess, "Kernel launch failed\n");
	
	err = hipMemcpyAsync(h_A, d_A, num_pkts * sizeof(int),
		hipMemcpyDeviceToHost, my_stream);
	CPE(err != hipSuccess, "D2H memcpy failed\n");

	hipStreamSynchronize(my_stream);
}

void gpu_run(int *h_A, int *d_A, int num_pkts, hipStream_t my_stream)
{
	int err = hipSuccess;
	struct timespec h2d_start[ITERS], h2d_end[ITERS];
	struct timespec kernel_start[ITERS], kernel_end[ITERS];
	struct timespec d2h_start[ITERS], d2h_end[ITERS];
	struct timespec sync_start[ITERS], sync_end[ITERS];

	/** < Microseconds */
	double h2d_diff[ITERS], kernel_diff[ITERS], d2h_diff[ITERS], sync_diff[ITERS];
	double h2d_tot = 0, kernel_tot = 0, d2h_tot = 0, sync_tot = 0;
	
	int i, j;
	int threadsPerBlock = 256;
	int blocksPerGrid = (num_pkts + threadsPerBlock - 1) / threadsPerBlock;

	/** < Do a dummy run for warmup */
	dummy_run(h_A, d_A, num_pkts, my_stream);

	/** < Run several iterations */
	for(i = 0; i < ITERS; i ++) {

		for(j = 0; j < num_pkts; j++)	{
			h_A[j] = i;
		}

		/** < Host-to-device memcpy */
		clock_gettime(CLOCK_REALTIME, &h2d_start[i]);
		err = hipMemcpyAsync(d_A, h_A, num_pkts * sizeof(int),
			hipMemcpyHostToDevice, my_stream);
		CPE(err != hipSuccess, "H2D memcpy failed\n");
		clock_gettime(CLOCK_REALTIME, &h2d_end[i]);

		/** < Kernel launch */
		clock_gettime(CLOCK_REALTIME, &kernel_start[i]);
		vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, my_stream>>>(d_A, num_pkts);
		clock_gettime(CLOCK_REALTIME, &kernel_end[i]);

		err = hipGetLastError();
		CPE(err != hipSuccess, "Kernel launch failed\n");

		/** < Device-to-host memcpy */
		clock_gettime(CLOCK_REALTIME, &d2h_start[i]);
		err = hipMemcpyAsync(h_A, d_A, num_pkts * sizeof(int),
			hipMemcpyDeviceToHost, my_stream);
		CPE(err != hipSuccess, "D2H memcpy failed\n");
		clock_gettime(CLOCK_REALTIME, &d2h_end[i]);

		/** < Wait for operation completion */
		clock_gettime(CLOCK_REALTIME, &sync_start[i]);
		hipStreamSynchronize(my_stream);
		clock_gettime(CLOCK_REALTIME, &sync_end[i]);

		/** < Measure the difference */
		h2d_diff[i] =
			(double) (h2d_end[i].tv_nsec - h2d_start[i].tv_nsec) / 1000 +
			(h2d_end[i].tv_sec - h2d_start[i].tv_sec) * 1000000;
		kernel_diff[i] =
			(double) (kernel_end[i].tv_nsec - kernel_start[i].tv_nsec) / 1000 +
			(kernel_end[i].tv_sec - kernel_start[i].tv_sec) * 1000000;
		d2h_diff[i] =
			(double) (d2h_end[i].tv_nsec - d2h_start[i].tv_nsec) / 1000 +
			(d2h_end[i].tv_sec - d2h_start[i].tv_sec) * 1000000;
		sync_diff[i] =
			(double) (sync_end[i].tv_nsec - sync_start[i].tv_nsec) / 1000 +
			(sync_end[i].tv_sec - sync_start[i].tv_sec) * 1000000;

		printf("ITER %d: h2d: %f us, kernel: %f us, d2h us: %f\n", i,
			h2d_diff[i], kernel_diff[i], d2h_diff[i]);

		h2d_tot += h2d_diff[i];
		kernel_tot += kernel_diff[i];
		d2h_tot += d2h_diff[i];
		sync_tot += sync_diff[i];

		/** < Check results */
		for(j = 0; j < num_pkts; j ++) {
			assert(h_A[j] == i * i);
		}
	}

	/** < Sort the times for percentiles */
	qsort(h2d_diff, ITERS, sizeof(double), cmpfunc);
	qsort(kernel_diff, ITERS, sizeof(double), cmpfunc);
	qsort(d2h_diff, ITERS, sizeof(double), cmpfunc);
	qsort(sync_diff, ITERS, sizeof(double), cmpfunc);

	int i_5 = (ITERS * 5) / 100;
	int i_95 = (ITERS * 5) / 100;

	red_printf("H2D average: %.2f us, 5th %.2f us, 95th: %.2f\n",
		h2d_tot / ITERS, h2d_diff[i_5], h2d_diff[i_95]);
	red_printf("Kernel average: %.2f us, 5th %.2f us, 95th: %.2f\n",
		kernel_tot / ITERS, kernel_diff[i_5], kernel_diff[i_95]);
	red_printf("D2H average: %.2f us, 5th %.2f us, 95th: %.2f\n",
		d2h_tot / ITERS, d2h_diff[i_5], d2h_diff[i_95]);
	red_printf("SYNC average: %.2f us, 5th %.2f us, 95th: %.2f\n",
		sync_tot / ITERS, sync_diff[i_5], sync_diff[i_95]);
	
	red_printf("TOT average %.2f us 5th %.2f us 95th %.2f\n",
		(h2d_tot + kernel_tot + d2h_tot + sync_tot) / ITERS,
		(h2d_diff[i_5] + kernel_diff[i_5] + d2h_diff[i_5] + sync_diff[i_5]),
		(d2h_diff[i_95] + kernel_diff[i_95] + d2h_diff[i_95] + sync_diff[i_95]));
}

int main(int argc, char *argv[])
{
	int err = hipSuccess;
	int *h_A, *d_A;
	hipStream_t my_stream;

	assert(argc == 2);
	int num_pkts = atoi(argv[1]);

	printDeviceProperties();

	/** < Create a CUDA stream for asynch operations */
	err = hipStreamCreate(&my_stream);
	CPE(err != hipSuccess, "Failed to create cudaStream\n");

	/** < Allocate host and device buffers */
	h_A = (int *) malloc(num_pkts * sizeof(int));
	err = hipMalloc((void **) &d_A, num_pkts * sizeof(int));
	CPE(err != hipSuccess, "Failed to hipMalloc\n");

	if (h_A == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	/** < Run the measurement code */
	gpu_run(h_A, d_A, num_pkts, my_stream);
	
	/** < Free host and device memory */
	free(h_A);
	hipFree(d_A);

	// Reset the device and exit
	err = hipDeviceReset();
	CPE(err != hipSuccess, "Failed to de-initialize the device\n");
	return 0;
}

